#include "hip/hip_runtime.h"
# // FILE: gpu-image-suite/src/gpu_filters.cu
#include <hip/hip_runtime.h>
#include <opencv2/imgproc.hpp>
#include "gpu_filters.hpp"
#include <vector>
#include <stdexcept>
static inline void checkCuda(hipError_t e){ if(e!=hipSuccess) throw std::runtime_error(hipGetErrorString(e)); }

// ===== Gaussian (separable) =====
__constant__ float d_kernel[64];

template<int R, int BW, int BH>
__global__ void blur_horiz(const unsigned char* in, unsigned char* out, int w, int h){
    __shared__ unsigned char tile[BH][BW + 2*R];
    int x = blockIdx.x*BW + threadIdx.x;
    int y = blockIdx.y*BH + threadIdx.y;
    int tx = threadIdx.x; int ty = threadIdx.y;
    for(int dx=tx; dx<BW+2*R; dx+=BW){
        int gx = blockIdx.x*BW + dx - R; gx = max(0, min(w-1, gx));
        if (y<h) tile[ty][dx] = in[y*w + gx];
    }
    __syncthreads();
    if (x<w && y<h){
        float acc=0.f;
        #pragma unroll
        for(int k=-R;k<=R;++k) acc += d_kernel[k+R] * tile[ty][tx + k + R];
        out[y*w + x] = (unsigned char)min(255.f, max(0.f, acc));
    }
}

template<int R, int BW, int BH>
__global__ void blur_vert(const unsigned char* in, unsigned char* out, int w, int h){
    __shared__ unsigned char tile[BH + 2*R][BW];
    int x = blockIdx.x*BW + threadIdx.x;
    int y = blockIdx.y*BH + threadIdx.y;
    int tx = threadIdx.x; int ty = threadIdx.y;
    for(int dy=ty; dy<BH+2*R; dy+=BH){
        int gy = blockIdx.y*BH + dy - R; gy = max(0, min(h-1, gy));
        if (x<w) tile[dy][tx] = in[gy*w + x];
    }
    __syncthreads();
    if (x<w && y<h){
        float acc=0.f;
        #pragma unroll
        for(int k=-R;k<=R;++k) acc += d_kernel[k+R] * tile[ty + k + R][tx];
        out[y*w + x] = (unsigned char)min(255.f, max(0.f, acc));
    }
}

static std::vector<float> gaussian_kernel(int radius, double sigma){
    int k = 2*radius+1; std::vector<float> h(k);
    double s2=2*sigma*sigma, sum=0;
    for(int i=-radius;i<=radius;++i){ double v = std::exp(-(i*i)/s2); h[i+radius]=(float)v; sum+=v; }
    for(auto& v: h) v = (float)(v/sum);
    return h;
}

// ===== Sobel 3x3 magnitude =====
__global__ void sobel3x3(const unsigned char* in, unsigned char* out, int w, int h){
    __shared__ unsigned char tile[16+2][16+2];
    int bx = blockIdx.x*16, by = blockIdx.y*16;
    int tx = threadIdx.x, ty = threadIdx.y;
    int x = bx + tx; int y = by + ty;
    int ix = min(max(x-1,0), w-1);
    int iy = min(max(y-1,0), h-1);
    tile[ty][tx] = in[iy*w + ix];
    __syncthreads();
    if (tx>=1 && tx<17 && ty>=1 && ty<17 && x<w && y<h){
        int gx = -tile[ty-1][tx-1]-2*tile[ty][tx-1]-tile[ty+1][tx-1]
                 +tile[ty-1][tx+1]+2*tile[ty][tx+1]+tile[ty+1][tx+1];
        int gy =  tile[ty-1][tx-1]+2*tile[ty-1][tx]+tile[ty-1][tx+1]
                 -tile[ty+1][tx-1]-2*tile[ty+1][tx]-tile[ty+1][tx+1];
        int mag = min(255, (int)sqrtf((float)(gx*gx + gy*gy)));
        out[y*w + x] = (unsigned char)mag;
    }
}

namespace gpu {

cv::Mat gaussian_blur(const cv::Mat& imgBGR, int radius, double sigma){
    if (imgBGR.empty()) throw std::runtime_error("Empty image");
    cv::Mat gray; cv::cvtColor(imgBGR, gray, cv::COLOR_BGR2GRAY);
    int w=gray.cols, h=gray.rows; size_t n=(size_t)w*h;
    uchar *d_in=nullptr, *d_tmp=nullptr, *d_out=nullptr;
    checkCuda(hipMalloc(&d_in, n));
    checkCuda(hipMalloc(&d_tmp, n));
    checkCuda(hipMalloc(&d_out, n));
    checkCuda(hipMemcpy(d_in, gray.data, n, hipMemcpyHostToDevice));
    auto hker = gaussian_kernel(radius, sigma);
    checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(d_kernel), hker.data(), hker.size()*sizeof(float)));
    dim3 block1(32,8);
    dim3 grid1((w + block1.x - 1)/block1.x, (h + block1.y - 1)/block1.y);
    switch(radius){
        case 3:
            blur_horiz<3,32,8><<<grid1, block1>>>(d_in, d_tmp, w, h);
            blur_vert <3,32,8><<<grid1, block1>>>(d_tmp, d_out, w, h);
            break;
        default:
            checkCuda(hipMemcpy(d_out, d_in, n, hipMemcpyDeviceToDevice));
            break;
    }
    checkCuda(hipGetLastError());
    cv::Mat out(h, w, CV_8U);
    checkCuda(hipMemcpy(out.data, d_out, n, hipMemcpyDeviceToHost));
    hipFree(d_in); hipFree(d_tmp); hipFree(d_out);
    return out;
}

cv::Mat sobel_edge(const cv::Mat& imgBGR){
    cv::Mat gray; cv::cvtColor(imgBGR, gray, cv::COLOR_BGR2GRAY);
    int w=gray.cols, h=gray.rows; size_t n=(size_t)w*h;
    uchar *d_in=nullptr, *d_out=nullptr;
    checkCuda(hipMalloc(&d_in, n));
    checkCuda(hipMalloc(&d_out, n));
    checkCuda(hipMemcpy(d_in, gray.data, n, hipMemcpyHostToDevice));
    dim3 block(18,18); dim3 grid((w+15)/16, (h+15)/16);
    sobel3x3<<<grid, block>>>(d_in, d_out, w, h);
    checkCuda(hipGetLastError());
    cv::Mat out(h,w,CV_8U);
    checkCuda(hipMemcpy(out.data, d_out, n, hipMemcpyDeviceToHost));
    hipFree(d_in); hipFree(d_out);
    return out;
}

cv::Mat sharpen_unsharp(const cv::Mat& imgBGR, int radius, double sigma, double amount){
    // Hybrid (uses GPU blur but combines on CPU for brevity)
    cv::Mat gray; cv::cvtColor(imgBGR, gray, cv::COLOR_BGR2GRAY);
    cv::Mat blur = gaussian_blur(imgBGR, radius, sigma);
    cv::Mat detail; cv::subtract(gray, blur, detail, cv::noArray(), CV_16S);
    cv::Mat out16; cv::addWeighted(gray, 1.0, detail, amount/255.0, 0.0, out16, CV_16S);
    cv::Mat out8; out16.convertTo(out8, CV_8U);
    return out8;
}

cv::Mat hist_equalize(const cv::Mat& imgBGR){
    // TODO: GPU histogram equalization
    cv::Mat ycrcb; cv::cvtColor(imgBGR, ycrcb, cv::COLOR_BGR2YCrCb);
    std::vector<cv::Mat> ch; cv::split(ycrcb, ch);
    cv::equalizeHist(ch[0], ch[0]);
    cv::merge(ch, ycrcb);
    cv::Mat out; cv::cvtColor(ycrcb, out, cv::COLOR_YCrCb2BGR);
    return out;
}
} // namespace gpu
